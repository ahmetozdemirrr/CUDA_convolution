#include "hip/hip_runtime.h"
/* cuda_convolution.cu */

#include <hip/hip_runtime.h>
#include <iostream>
#include <vector>
#include <cmath>
#include "lodepng.h"

#define CHECK_CUDA_ERROR(call)                                           \
    {                                                                    \
        const hipError_t error = call;                                  \
        if (error != hipSuccess) {                                      \
            std::cerr << "Error: " << __FILE__ << ", line " << __LINE__  \
                      << ": " << hipGetErrorString(error) << std::endl; \
            exit(1);                                                     \
        }                                                                \
    }


const int KERNEL_SIZE = 3;
__constant__ float d_kernel[KERNEL_SIZE * KERNEL_SIZE]; /* Convolution kernel in constant memory */


/* CUDA kernel without shared memory */
__global__ void convolutionGlobal(const unsigned char * input, unsigned char * output, int width, int height) 
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) 
    {
        float sum = 0.0f;

        for (int ky = -1; ky <= 1; ++ky) 
        {
            for (int kx = -1; kx <= 1; ++kx) 
            {
                int ix = min(max(x + kx, 0), width - 1);
                int iy = min(max(y + ky, 0), height - 1);
                sum += input[iy * width + ix] * d_kernel[(ky + 1) * KERNEL_SIZE + (kx + 1)];
            }
        }
        output[y * width + x] = static_cast<unsigned char>(fminf(fmaxf(sum, 0.0f), 255.0f));
    }
}

/* CUDA kernel with shared memory */
__global__ void convolutionShared(const unsigned char * input, unsigned char * output, int width, int height) 
{
    extern __shared__ unsigned char s_input[];

    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    int lx = threadIdx.x + 1;
    int ly = threadIdx.y + 1;

    int local_width = blockDim.x + 2;

    /* Load input into shared memory */
    if (x < width && y < height) 
    {
        s_input[ly * local_width + lx] = input[y * width + x];
    }

    if (threadIdx.x == 0 && x > 0)
    {
        s_input[ly * local_width] = input[y * width + (x - 1)];
    }
    
    else if (threadIdx.x == blockDim.x - 1 && x < width - 1)
    {
        s_input[ly * local_width + lx + 1] = input[y * width + (x + 1)];
    }

    if (threadIdx.y == 0 && y > 0)
    {
        s_input[(ly - 1) * local_width + lx] = input[(y - 1) * width + x];
    }
    
    else if (threadIdx.y == blockDim.y - 1 && y < height - 1)
    {
        s_input[(ly + 1) * local_width + lx] = input[(y + 1) * width + x];
    }
    __syncthreads();

    /* Perform convolution */
    if (x < width && y < height) 
    {
        float sum = 0.0f;

        for (int ky = -1; ky <= 1; ++ky) 
        {
            for (int kx = -1; kx <= 1; ++kx) 
            {
                sum += s_input[(ly + ky) * local_width + (lx + kx)] * d_kernel[(ky + 1) * KERNEL_SIZE + (kx + 1)];
            }
        }
        output[y * width + x] = static_cast<unsigned char>(fminf(fmaxf(sum, 0.0f), 255.0f));
    }
}

/* Load image using lodepng */
std::vector<unsigned char> loadImage(const char * filename, unsigned int & width, unsigned int & height) 
{
    std::vector<unsigned char> image;
    unsigned error = lodepng::decode(image, width, height, filename);

    if (error) 
    {
        std::cerr << "Error loading image: " << lodepng_error_text(error) << std::endl;
        exit(1);
    }
    return image;
}

/* Save image using lodepng */
void saveImage(const char * filename, const std::vector<unsigned char> & image, unsigned int width, unsigned int height) 
{
    if (image.size() != width * height) 
    {
        std::cerr << "Error: Image size mismatch with dimensions: "
                  << "Expected " << (width * height) << " but got " << image.size() << std::endl;
        return;
    }
    std::vector<unsigned char> rgbaImage(width * height * 4, 255);

    for (size_t i = 0; i < width * height; ++i) 
    {
        rgbaImage[4 * i + 0] = image[i];
        rgbaImage[4 * i + 1] = image[i];
        rgbaImage[4 * i + 2] = image[i];
    }
    unsigned error = lodepng::encode(filename, rgbaImage, width, height);

    if (error) 
    {
        std::cerr << "Error saving image: " << lodepng_error_text(error) << std::endl;
        exit(1);
    }
}

void printSharedMemoryLimit() 
{
    hipDeviceProp_t prop;
    int device;

    CHECK_CUDA_ERROR(hipGetDevice(&device));
    CHECK_CUDA_ERROR(hipGetDeviceProperties(&prop, device));

    std::cout << "Shared Memory Per Block: " << prop.sharedMemPerBlock << " bytes" << std::endl;
}

int main(int argc, char ** argv) 
{
    if (argc < 7) 
    {
        std::cerr << "Usage: " << argv[0] << " <input.png> <output.png> <mode (0=global, 1=shared)> <block_size> <grid_x> <grid_y>" << std::endl;
        return 1;
    }
    printSharedMemoryLimit();  /* Print shared memory limit for Python script */

    const char * inputFile  = argv[1];
    const char * outputFile = argv[2];

    int mode      = atoi(argv[3]);
    int blockSize = atoi(argv[4]);
    int gridX     = atoi(argv[5]);
    int gridY     = atoi(argv[6]);

    unsigned int width, height;
    auto hostInput = loadImage(inputFile, width, height);

    if (hostInput.size() != width * height * 4) 
    {
        std::cerr << "Error: Input image dimensions do not match expected size." << std::endl;
        return 1;
    }

    /* Grayscale conversion */
    size_t imageSize = width * height;
    std::vector<unsigned char> grayscaleInput(imageSize);

    for (size_t i = 0; i < imageSize; ++i) 
    {
        unsigned char r = hostInput[4 * i + 0];
        unsigned char g = hostInput[4 * i + 1];
        unsigned char b = hostInput[4 * i + 2];
        grayscaleInput[i] = static_cast<unsigned char>(0.299f * r + 0.587f * g + 0.114f * b);
    }
    std::vector<unsigned char> hostOutput(imageSize);
    unsigned char *d_input, *d_output;

    CHECK_CUDA_ERROR(hipMalloc(&d_input, imageSize));
    CHECK_CUDA_ERROR(hipMalloc(&d_output, imageSize));

    float h_kernel[KERNEL_SIZE * KERNEL_SIZE] = 
    {
        -1, -1, -1,
        -1,  8, -1,
        -1, -1, -1
    };
    CHECK_CUDA_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(d_kernel), h_kernel, sizeof(h_kernel)));
    CHECK_CUDA_ERROR(hipMemcpy(d_input, grayscaleInput.data(), imageSize, hipMemcpyHostToDevice));

    dim3 block(blockSize, blockSize);
    dim3 grid(gridX, gridY);

    if (mode == 0) 
    {
        convolutionGlobal<<<grid, block>>>(d_input, d_output, width, height);
    } 

    else if (mode == 1) 
    {
        size_t sharedMemSize = (block.x + 2) * (block.y + 2) * sizeof(unsigned char);
        convolutionShared<<<grid, block, sharedMemSize>>>(d_input, d_output, width, height);
    } 

    else 
    {
        std::cerr << "Invalid mode. Use 0 for global memory or 1 for shared memory." << std::endl;
        return 1;
    }
    CHECK_CUDA_ERROR(hipDeviceSynchronize());
    CHECK_CUDA_ERROR(hipMemcpy(hostOutput.data(), d_output, imageSize, hipMemcpyDeviceToHost));
    
    saveImage(outputFile, hostOutput, width, height);

    hipFree(d_input);
    hipFree(d_output);

    return 0;
}
